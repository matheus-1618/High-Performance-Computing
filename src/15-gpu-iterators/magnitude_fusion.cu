#include "hip/hip_runtime.h"
#include <iostream>
 #include <iomanip>
 #include <cstdlib>
 #include <chrono>
 #include <cstdlib>
 #include <algorithm>
 #include <thrust/host_vector.h>
 #include <thrust/device_vector.h>
 #include <thrust/generate.h>
 #include <thrust/functional.h>
 #include <thrust/copy.h>
 #include <cmath>
 #include <random>

 using namespace std::chrono;

 void reportTime(const char* msg, steady_clock::duration span) {
     auto ms = duration_cast<milliseconds>(span);
     std::cout << msg << " - levou - " <<
      ms.count() << " milisegundos" << std::endl;
 }

 // CRIE UMA FUNCTOR PARA CALCULAR A SQUARE
struct square {
    __host__ __device__
    float operator()(const float &x) const {
        return x * x;
    }
};


 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
 float magnitude(thrust::device_vector<float> x) {
     float result;
     thrust::transform(x.begin() , x.end() , x.begin() , square());  // se não colocar nada ou só o begin ele sobrescreve o resultado no próprio vetor   
      result = std::sqrt(thrust::transform_reduce(x.begin(), x.end(), square(), 0.0f, thrust::plus<float>()));
     return result;

 }

 int main(int argc, char** argv) {
     if (argc != 2) {
         std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
         std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
         return 1;
     }
     int n = std::atoi(argv[1]); //numero de elementos
     steady_clock::time_point ts, te;

     // Faça um  vector em thrust 
     thrust::host_vector<float> vetor(n);


     // inicilize o  vector
     ts = steady_clock::now();

     std::generate(vetor.begin(),vetor.end(), std::rand);


     te = steady_clock::now();
     reportTime("Inicializacao", te - ts);

     //Transferindo para gpu
     thrust::device_vector<float> v_d = vetor;

     // Calcule a magnitude do vetor
     ts = steady_clock::now();
     float len = magnitude(v_d);
     te = steady_clock::now();
     reportTime("Tempo para calculo", te - ts);


     std::cout << std::fixed << std::setprecision(4);
     std::cout << "Magnitude : " << len << std::endl;
 }
